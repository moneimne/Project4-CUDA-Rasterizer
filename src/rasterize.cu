#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya, Shuai Shao (Shrek)
 * @date      2012-2016
 * @copyright University of Pennsylvania & STUDENT
 */

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include <util/tiny_gltf_loader.h>
#include "rasterizeTools.h"
#include "rasterize.h"
#include <glm/gtc/quaternion.hpp>
#include <glm/gtc/matrix_transform.hpp>

//#define BILINEAR_FILTERING
//#define PERSPECTIVE_CORRECT_TEXTURE
//#define GAUSSIAN_BLUR

namespace {

	typedef unsigned short VertexIndex;
	typedef glm::vec3 VertexAttributePosition;
	typedef glm::vec3 VertexAttributeNormal;
	typedef glm::vec2 VertexAttributeTexcoord;
	typedef unsigned char TextureData;

	typedef unsigned char BufferByte;

	enum PrimitiveType{
		Point = 1,
		Line = 2,
		Triangle = 3
	};

	struct VertexOut {
		glm::vec4 pos;

		// TODO: add new attributes to your VertexOut
		// The attributes listed below might be useful, 
		// but always feel free to modify on your own

		 glm::vec3 eyePos;	// eye space position used for shading
		 glm::vec3 eyeNor;	// eye space normal used for shading, cuz normal will go wrong after perspective transformation
		// glm::vec3 col;
		 glm::vec2 texcoord0;
		 TextureData* dev_diffuseTex = NULL;
		 int texWidth, texHeight;
		// ...
	};

	struct Primitive {
		PrimitiveType primitiveType = Triangle;	// C++ 11 init
		VertexOut v[3];
	};

	struct Fragment {
		glm::vec3 color;

		// TODO: add new attributes to your Fragment
		// The attributes listed below might be useful, 
		// but always feel free to modify on your own

		glm::vec3 eyePos;	// eye space position used for shading
		glm::vec3 eyeNor;
		VertexAttributeTexcoord texcoord0;
		TextureData* dev_diffuseTex;
		int texWidth, texHeight;
		// ...
	};

	struct PrimitiveDevBufPointers {
		int primitiveMode;	//from tinygltfloader macro
		PrimitiveType primitiveType;
		int numPrimitives;
		int numIndices;
		int numVertices;

		// Vertex In, const after loaded
		VertexIndex* dev_indices;
		VertexAttributePosition* dev_position;
		VertexAttributeNormal* dev_normal;
		VertexAttributeTexcoord* dev_texcoord0;

		// Materials, add more attributes when needed
		TextureData* dev_diffuseTex;
		int diffuseTexWidth;
		int diffuseTexHeight;
		// TextureData* dev_specularTex;
		// TextureData* dev_normalTex;
		// ...

		// Vertex Out, vertex used for rasterization, this is changing every frame
		VertexOut* dev_verticesOut;

		// TODO: add more attributes when needed
	};

}

static std::map<std::string, std::vector<PrimitiveDevBufPointers>> mesh2PrimitivesMap;


static int width = 0;
static int height = 0;

static int totalNumPrimitives = 0;
static Primitive *dev_primitives = NULL;
static Fragment *dev_fragmentBuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;
static glm::vec3 *dev_postprocess = NULL;

static int *dev_mutex = NULL;

static float * dev_depth = NULL;	// you might need this buffer when doing depth test

/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__ 
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

__device__
glm::vec3 colorAt(TextureData* texture, int textureWidth, float u, float v) {
	int flatIndex = u + v * textureWidth;
	float r = (float)texture[flatIndex * 3] / 255.0f;
	float g = (float)texture[flatIndex * 3 + 1] / 255.0f;
	float b = (float)texture[flatIndex * 3 + 2] / 255.0f;
	return glm::vec3(r, g, b);
}

__device__
glm::vec3 getBilinearFilteredColor(TextureData* texture, int textureWidth, int textureHeight, float u, float v) {
	float x = u * (float)textureWidth;
	float y = v * (float)textureHeight;
	float floorX = glm::floor(x);
	float floorY = glm::floor(y);
	float deltaX = x - floorX;
	float deltaY = y - floorY;

	int xPos = (int)floorX;
	int yPos = (int)floorY;
	int xPlusOne = glm::clamp(xPos + 1, 0, textureWidth - 1);
	int yPlusOne = glm::clamp(yPos + 1, 0, textureHeight - 1);

	glm::vec3 v0 = colorAt(texture, textureWidth, xPos, yPos);
	glm::vec3 v1 = colorAt(texture, textureWidth, xPlusOne, yPos);
	glm::vec3 v2 = colorAt(texture, textureWidth, xPos, yPlusOne);
	glm::vec3 v3 = colorAt(texture, textureWidth, xPlusOne, yPlusOne);
	glm::vec3 mix01 = glm::mix(v0, v1, deltaX);
	glm::vec3 mix23 = glm::mix(v2, v3, deltaX);
	return glm::mix(mix01, mix23, deltaY);
}

/** 
* Writes fragment colors to the framebuffer
*/
__global__
void render(int w, int h, Fragment *fragmentBuffer, glm::vec3 *framebuffer, float *dev_depth) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
		Fragment fragment = fragmentBuffer[index];
		glm::vec3 normal = glm::normalize(fragment.eyeNor);
		glm::vec3 lightPos(0.0f);
		glm::vec3 lightDir = glm::normalize(lightPos - fragment.eyePos);

		glm::vec3 color;
		if (fragment.dev_diffuseTex != NULL) {
#ifdef BILINEAR_FILTERING
			color = getBilinearFilteredColor(fragment.dev_diffuseTex, fragment.texWidth, fragment.texHeight, fragment.texcoord0.x, fragment.texcoord0.y);
#else
			int u = fragment.texcoord0.x * fragment.texWidth;
			int v = fragment.texcoord0.y * fragment.texHeight;
			color = colorAt(fragment.dev_diffuseTex, fragment.texWidth, u, v);
#endif
		}
		else {
			color = fragment.color;
		}
		float NdotL = glm::dot(normal, lightDir);
		if (NdotL == NdotL) {
			framebuffer[index] = color * glm::dot(normal, lightDir);
		}
		else {
			framebuffer[index] = color;
		}
    }
}

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;
	hipFree(dev_fragmentBuffer);
	hipMalloc(&dev_fragmentBuffer, width * height * sizeof(Fragment));
	hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));
    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));
	hipFree(dev_postprocess);
	hipMalloc(&dev_postprocess, width * height * sizeof(glm::vec3));

	hipFree(dev_mutex);
	hipMalloc(&dev_mutex, width * height * sizeof(int));
	hipMemset(dev_mutex, 0, sizeof(int));
    
	hipFree(dev_depth);
	hipMalloc(&dev_depth, width * height * sizeof(int));

	checkCUDAError("rasterizeInit");
}

__global__
void initDepth(int w, int h, float * depth)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < w && y < h)
	{
		int index = x + (y * w);
		depth[index] = 1000000.0f;
	}
}


/**
* kern function with support for stride to sometimes replace hipMemcpy
* One thread is responsible for copying one component
*/
__global__ 
void _deviceBufferCopy(int N, BufferByte* dev_dst, const BufferByte* dev_src, int n, int byteStride, int byteOffset, int componentTypeByteSize) {
	
	// Attribute (vec3 position)
	// component (3 * float)
	// byte (4 * byte)

	// id of component
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < N) {
		int count = i / n;
		int offset = i - count * n;	// which component of the attribute

		for (int j = 0; j < componentTypeByteSize; j++) {
			
			dev_dst[count * componentTypeByteSize * n 
				+ offset * componentTypeByteSize 
				+ j]

				= 

			dev_src[byteOffset 
				+ count * (byteStride == 0 ? componentTypeByteSize * n : byteStride) 
				+ offset * componentTypeByteSize 
				+ j];
		}
	}
	

}

__global__
void _nodeMatrixTransform(
	int numVertices,
	VertexAttributePosition* position,
	VertexAttributeNormal* normal,
	glm::mat4 MV, glm::mat3 MV_normal) {

	// vertex id
	int vid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (vid < numVertices) {
		position[vid] = glm::vec3(MV * glm::vec4(position[vid], 1.0f));
		normal[vid] = glm::normalize(MV_normal * normal[vid]);
	}
}

glm::mat4 getMatrixFromNodeMatrixVector(const tinygltf::Node & n) {
	
	glm::mat4 curMatrix(1.0);

	const std::vector<double> &m = n.matrix;
	if (m.size() > 0) {
		// matrix, copy it

		for (int i = 0; i < 4; i++) {
			for (int j = 0; j < 4; j++) {
				curMatrix[i][j] = (float)m.at(4 * i + j);
			}
		}
	} else {
		// no matrix, use rotation, scale, translation

		if (n.translation.size() > 0) {
			curMatrix[3][0] = n.translation[0];
			curMatrix[3][1] = n.translation[1];
			curMatrix[3][2] = n.translation[2];
		}

		if (n.rotation.size() > 0) {
			glm::mat4 R;
			glm::quat q;
			q[0] = n.rotation[0];
			q[1] = n.rotation[1];
			q[2] = n.rotation[2];

			R = glm::mat4_cast(q);
			curMatrix = curMatrix * R;
		}

		if (n.scale.size() > 0) {
			curMatrix = curMatrix * glm::scale(glm::vec3(n.scale[0], n.scale[1], n.scale[2]));
		}
	}

	return curMatrix;
}

void traverseNode (
	std::map<std::string, glm::mat4> & n2m,
	const tinygltf::Scene & scene,
	const std::string & nodeString,
	const glm::mat4 & parentMatrix
	) 
{
	const tinygltf::Node & n = scene.nodes.at(nodeString);
	glm::mat4 M = parentMatrix * getMatrixFromNodeMatrixVector(n);
	n2m.insert(std::pair<std::string, glm::mat4>(nodeString, M));

	auto it = n.children.begin();
	auto itEnd = n.children.end();

	for (; it != itEnd; ++it) {
		traverseNode(n2m, scene, *it, M);
	}
}

void rasterizeSetBuffers(const tinygltf::Scene & scene) {

	totalNumPrimitives = 0;

	std::map<std::string, BufferByte*> bufferViewDevPointers;

	// 1. copy all `bufferViews` to device memory
	{
		std::map<std::string, tinygltf::BufferView>::const_iterator it(
			scene.bufferViews.begin());
		std::map<std::string, tinygltf::BufferView>::const_iterator itEnd(
			scene.bufferViews.end());

		for (; it != itEnd; it++) {
			const std::string key = it->first;
			const tinygltf::BufferView &bufferView = it->second;
			if (bufferView.target == 0) {
				continue; // Unsupported bufferView.
			}

			const tinygltf::Buffer &buffer = scene.buffers.at(bufferView.buffer);

			BufferByte* dev_bufferView;
			hipMalloc(&dev_bufferView, bufferView.byteLength);
			hipMemcpy(dev_bufferView, &buffer.data.front() + bufferView.byteOffset, bufferView.byteLength, hipMemcpyHostToDevice);

			checkCUDAError("Set BufferView Device Mem");

			bufferViewDevPointers.insert(std::make_pair(key, dev_bufferView));

		}
	}



	// 2. for each mesh: 
	//		for each primitive: 
	//			build device buffer of indices, material, and each attributes
	//			and store these pointers in a map
	{

		std::map<std::string, glm::mat4> nodeString2Matrix;
		auto rootNodeNamesList = scene.scenes.at(scene.defaultScene);

		{
			auto it = rootNodeNamesList.begin();
			auto itEnd = rootNodeNamesList.end();
			for (; it != itEnd; ++it) {
				traverseNode(nodeString2Matrix, scene, *it, glm::mat4(1.0f));
			}
		}


		// parse through node to access mesh

		auto itNode = nodeString2Matrix.begin();
		auto itEndNode = nodeString2Matrix.end();
		for (; itNode != itEndNode; ++itNode) {

			const tinygltf::Node & N = scene.nodes.at(itNode->first);
			const glm::mat4 & matrix = itNode->second;
			const glm::mat3 & matrixNormal = glm::transpose(glm::inverse(glm::mat3(matrix)));

			auto itMeshName = N.meshes.begin();
			auto itEndMeshName = N.meshes.end();

			for (; itMeshName != itEndMeshName; ++itMeshName) {

				const tinygltf::Mesh & mesh = scene.meshes.at(*itMeshName);

				auto res = mesh2PrimitivesMap.insert(std::pair<std::string, std::vector<PrimitiveDevBufPointers>>(mesh.name, std::vector<PrimitiveDevBufPointers>()));
				std::vector<PrimitiveDevBufPointers> & primitiveVector = (res.first)->second;

				// for each primitive
				for (size_t i = 0; i < mesh.primitives.size(); i++) {
					const tinygltf::Primitive &primitive = mesh.primitives[i];

					if (primitive.indices.empty())
						return;

					// TODO: add new attributes for your PrimitiveDevBufPointers when you add new attributes
					VertexIndex* dev_indices = NULL;
					VertexAttributePosition* dev_position = NULL;
					VertexAttributeNormal* dev_normal = NULL;
					VertexAttributeTexcoord* dev_texcoord0 = NULL;

					// ----------Indices-------------

					const tinygltf::Accessor &indexAccessor = scene.accessors.at(primitive.indices);
					const tinygltf::BufferView &bufferView = scene.bufferViews.at(indexAccessor.bufferView);
					BufferByte* dev_bufferView = bufferViewDevPointers.at(indexAccessor.bufferView);

					// assume type is SCALAR for indices
					int n = 1;
					int numIndices = indexAccessor.count;
					int componentTypeByteSize = sizeof(VertexIndex);
					int byteLength = numIndices * n * componentTypeByteSize;

					dim3 numThreadsPerBlock(128);
					dim3 numBlocks((numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
					hipMalloc(&dev_indices, byteLength);
					_deviceBufferCopy << <numBlocks, numThreadsPerBlock >> > (
						numIndices,
						(BufferByte*)dev_indices,
						dev_bufferView,
						n,
						indexAccessor.byteStride,
						indexAccessor.byteOffset,
						componentTypeByteSize);


					checkCUDAError("Set Index Buffer");


					// ---------Primitive Info-------

					// Warning: LINE_STRIP is not supported in tinygltfloader
					int numPrimitives;
					PrimitiveType primitiveType;
					switch (primitive.mode) {
					case TINYGLTF_MODE_TRIANGLES:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices / 3;
						break;
					case TINYGLTF_MODE_TRIANGLE_STRIP:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices - 2;
						break;
					case TINYGLTF_MODE_TRIANGLE_FAN:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices - 2;
						break;
					case TINYGLTF_MODE_LINE:
						primitiveType = PrimitiveType::Line;
						numPrimitives = numIndices / 2;
						break;
					case TINYGLTF_MODE_LINE_LOOP:
						primitiveType = PrimitiveType::Line;
						numPrimitives = numIndices + 1;
						break;
					case TINYGLTF_MODE_POINTS:
						primitiveType = PrimitiveType::Point;
						numPrimitives = numIndices;
						break;
					default:
						// output error
						break;
					};


					// ----------Attributes-------------

					auto it(primitive.attributes.begin());
					auto itEnd(primitive.attributes.end());

					int numVertices = 0;
					// for each attribute
					for (; it != itEnd; it++) {
						const tinygltf::Accessor &accessor = scene.accessors.at(it->second);
						const tinygltf::BufferView &bufferView = scene.bufferViews.at(accessor.bufferView);

						int n = 1;
						if (accessor.type == TINYGLTF_TYPE_SCALAR) {
							n = 1;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC2) {
							n = 2;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC3) {
							n = 3;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC4) {
							n = 4;
						}

						BufferByte * dev_bufferView = bufferViewDevPointers.at(accessor.bufferView);
						BufferByte ** dev_attribute = NULL;

						numVertices = accessor.count;
						int componentTypeByteSize;

						// Note: since the type of our attribute array (dev_position) is static (float32)
						// We assume the glTF model attribute type are 5126(FLOAT) here

						if (it->first.compare("POSITION") == 0) {
							componentTypeByteSize = sizeof(VertexAttributePosition) / n;
							dev_attribute = (BufferByte**)&dev_position;
						}
						else if (it->first.compare("NORMAL") == 0) {
							componentTypeByteSize = sizeof(VertexAttributeNormal) / n;
							dev_attribute = (BufferByte**)&dev_normal;
						}
						else if (it->first.compare("TEXCOORD_0") == 0) {
							componentTypeByteSize = sizeof(VertexAttributeTexcoord) / n;
							dev_attribute = (BufferByte**)&dev_texcoord0;
						}

						std::cout << accessor.bufferView << "  -  " << it->second << "  -  " << it->first << '\n';

						dim3 numThreadsPerBlock(128);
						dim3 numBlocks((n * numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
						int byteLength = numVertices * n * componentTypeByteSize;
						hipMalloc(dev_attribute, byteLength);

						_deviceBufferCopy << <numBlocks, numThreadsPerBlock >> > (
							n * numVertices,
							*dev_attribute,
							dev_bufferView,
							n,
							accessor.byteStride,
							accessor.byteOffset,
							componentTypeByteSize);

						std::string msg = "Set Attribute Buffer: " + it->first;
						checkCUDAError(msg.c_str());
					}

					// malloc for VertexOut
					VertexOut* dev_vertexOut;
					hipMalloc(&dev_vertexOut, numVertices * sizeof(VertexOut));
					checkCUDAError("Malloc VertexOut Buffer");

					// ----------Materials-------------

					// You can only worry about this part once you started to 
					// implement textures for your rasterizer
					TextureData* dev_diffuseTex = NULL;
					int diffuseTexWidth = 0;
					int diffuseTexHeight = 0;
					if (!primitive.material.empty()) {
						const tinygltf::Material &mat = scene.materials.at(primitive.material);
						printf("material.name = %s\n", mat.name.c_str());

						if (mat.values.find("diffuse") != mat.values.end()) {
							std::string diffuseTexName = mat.values.at("diffuse").string_value;
							if (scene.textures.find(diffuseTexName) != scene.textures.end()) {
								const tinygltf::Texture &tex = scene.textures.at(diffuseTexName);
								if (scene.images.find(tex.source) != scene.images.end()) {
									const tinygltf::Image &image = scene.images.at(tex.source);

									size_t s = image.image.size() * sizeof(TextureData);
									hipMalloc(&dev_diffuseTex, s);
									hipMemcpy(dev_diffuseTex, &image.image.at(0), s, hipMemcpyHostToDevice);
									
									diffuseTexWidth = image.width;
									diffuseTexHeight = image.height;

									checkCUDAError("Set Texture Image data");
								}
							}
						}

						// TODO: write your code for other materails
						// You may have to take a look at tinygltfloader
						// You can also use the above code loading diffuse material as a start point 
					}


					// ---------Node hierarchy transform--------
					hipDeviceSynchronize();
					
					dim3 numBlocksNodeTransform((numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
					_nodeMatrixTransform << <numBlocksNodeTransform, numThreadsPerBlock >> > (
						numVertices,
						dev_position,
						dev_normal,
						matrix,
						matrixNormal);

					checkCUDAError("Node hierarchy transformation");

					// at the end of the for loop of primitive
					// push dev pointers to map
					primitiveVector.push_back(PrimitiveDevBufPointers{
						primitive.mode,
						primitiveType,
						numPrimitives,
						numIndices,
						numVertices,

						dev_indices,
						dev_position,
						dev_normal,
						dev_texcoord0,

						dev_diffuseTex,
						diffuseTexWidth,
						diffuseTexHeight,

						dev_vertexOut	//VertexOut
					});

					totalNumPrimitives += numPrimitives;

				} // for each primitive

			} // for each mesh

		} // for each node

	}
	

	// 3. Malloc for dev_primitives
	{
		hipMalloc(&dev_primitives, totalNumPrimitives * sizeof(Primitive));
	}
	

	// Finally, hipFree raw dev_bufferViews
	{

		std::map<std::string, BufferByte*>::const_iterator it(bufferViewDevPointers.begin());
		std::map<std::string, BufferByte*>::const_iterator itEnd(bufferViewDevPointers.end());
			
			//bufferViewDevPointers

		for (; it != itEnd; it++) {
			hipFree(it->second);
		}

		checkCUDAError("Free BufferView Device Mem");
	}


}



__global__ 
void _vertexTransformAndAssembly(
	int numVertices, 
	PrimitiveDevBufPointers primitive, 
	glm::mat4 MVP, glm::mat4 MV, glm::mat3 MV_normal, 
	int width, int height) {

	// vertex id
	int vid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (vid < numVertices) {

		// TODO: Apply vertex transformation here
		// Multiply the MVP matrix for each vertex position, this will transform everything into clipping space
		glm::vec4 modelPosition(primitive.dev_position[vid], 1.0f);
		glm::vec4 clipPosition = MVP * modelPosition;
		// Then divide the pos by its w element to transform into NDC space
		glm::vec4 ndcPosition = clipPosition / clipPosition.w;
		// Finally transform x and y to viewport space
		float x = (ndcPosition.x + 1.0f) * (float)width * 0.5f;
		float y = (1.0f - ndcPosition.y) * (float)height * 0.5f;
		float z = -ndcPosition.z;

		// TODO: Apply vertex assembly here
		// Assemble all attribute arrays into the primitive array
		primitive.dev_verticesOut[vid].pos = glm::vec4(x, y, z, 1.0f);
		primitive.dev_verticesOut[vid].eyePos = glm::vec3(MV * modelPosition);
		primitive.dev_verticesOut[vid].eyeNor = glm::normalize(MV_normal * primitive.dev_normal[vid]);
		primitive.dev_verticesOut[vid].texcoord0 = primitive.dev_texcoord0[vid];
		primitive.dev_verticesOut[vid].dev_diffuseTex = primitive.dev_diffuseTex;
		primitive.dev_verticesOut[vid].texWidth = primitive.diffuseTexWidth;
		primitive.dev_verticesOut[vid].texHeight = primitive.diffuseTexHeight;
	}
}



static int curPrimitiveBeginId = 0;

__global__ 
void _primitiveAssembly(int numIndices, int curPrimitiveBeginId, Primitive* dev_primitives, PrimitiveDevBufPointers primitive) {

	// index id
	int iid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (iid < numIndices) {

		// TODO: uncomment the following code for a start
		// This is primitive assembly for triangles

		int pid;	// id for cur primitives vector
		if (primitive.primitiveMode == TINYGLTF_MODE_TRIANGLES) {
			pid = iid / (int)primitive.primitiveType;
			dev_primitives[pid + curPrimitiveBeginId].v[iid % (int)primitive.primitiveType]
				= primitive.dev_verticesOut[primitive.dev_indices[iid]];
		}


		// TODO: other primitive types (point, line)
	}
	
}

__global__
void _rasterize(int numPrimitives, Primitive* dev_primitives, Fragment* dev_fragmentBuffer, float* dev_depth, int* dev_mutex, int width, int height) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < numPrimitives) {
		Primitive primitive = dev_primitives[index];
		VertexOut v0 = primitive.v[0];
		VertexOut v1 = primitive.v[1];
		VertexOut v2 = primitive.v[2];

		glm::vec3 p0 = glm::vec3(v0.pos);
		glm::vec3 p1 = glm::vec3(v1.pos);
		glm::vec3 p2 = glm::vec3(v2.pos);
		glm::vec3 triPos[3] = { p0, p1, p2 };
		
		int minX = glm::max((int)glm::floor(glm::min(glm::min(p0.x, p1.x), p2.x)), 0);
		int minY = glm::max((int)glm::floor(glm::min(glm::min(p0.y, p1.y), p2.y)), 0);
		int maxX = glm::min((int)glm::ceil(glm::max(glm::max(p0.x, p1.x), p2.x)), width - 1);
		int maxY = glm::min((int)glm::ceil(glm::max(glm::max(p0.y, p1.y), p2.y)), height - 1);

		for (int x = minX; x <= maxX; x++) {
			for (int y = minY; y <= maxY; y++) {
				glm::vec2 p(x, y);
				// Position
				glm::vec3 barycentricCoord = calculateBarycentricCoordinate(triPos, p);
				if (isBarycentricCoordInBounds(barycentricCoord)) {
					int flatIndex = x + width * y;

					// Depth
					float depth = getZAtCoordinate(barycentricCoord, triPos);

					// Normal
					glm::vec3 normal = barycentricCoord.x * v0.eyeNor + barycentricCoord.y * v1.eyeNor + barycentricCoord.z * v2.eyeNor;

					// Position
					glm::vec3 position = barycentricCoord.x * v0.eyePos + barycentricCoord.y * v1.eyePos + barycentricCoord.z * v2.eyePos;

					// Texture coordinate
					glm::vec2 texCoord;
#ifdef PERSPECTIVE_CORRECT_TEXTURE
					glm::vec2 texCoordZ = barycentricCoord.x * (v0.texcoord0 / v0.eyePos.z) + barycentricCoord.y * (v1.texcoord0 / v1.eyePos.z) + barycentricCoord.z * (v2.texcoord0 / v2.eyePos.z);
					float coordZ = barycentricCoord.x * (1.0f / v0.eyePos.z) + barycentricCoord.y * (1.0f / v1.eyePos.z) + barycentricCoord.z * (1.0f / v2.eyePos.z);
					texCoord = texCoordZ / coordZ;
#else
					texCoord = barycentricCoord.x * v0.texcoord0 + barycentricCoord.y * v1.texcoord0 + barycentricCoord.z * v2.texcoord0;
#endif
					Fragment fragment;
					fragment.color = glm::vec3(0.0f, 0.0f, 0.8f);
					fragment.eyeNor = normal;
					fragment.eyePos = position;
					fragment.texcoord0 = texCoord;
					fragment.dev_diffuseTex = v0.dev_diffuseTex;
					fragment.texWidth = v0.texWidth;
					fragment.texHeight = v0.texHeight;
					bool isSet;
					do {
						isSet = (atomicCAS(&dev_mutex[flatIndex], 0, 1) == 0);
						if (isSet) {
							if (dev_depth[flatIndex] > depth) {
								dev_depth[flatIndex] = depth;
								dev_fragmentBuffer[flatIndex] = fragment;
							}

							// Reset mutex
							dev_mutex[flatIndex] = 0;
						}
					} while (!isSet);
				}
			}
		}
	}
}

__global__
void gaussianBlurWidth(int width, int height, glm::vec3 *dev_framebuffer, glm::vec3 *dev_postprocess) {
	extern __shared__ glm::vec3 sharedRow[];
	int x = threadIdx.x;
	int y = blockIdx.x;

	float weight[5] = { 0.227027f, 0.1945946, 0.1216216, 0.054054, 0.016216 };

	if (x >= width || y >= height) {
		return;
	}

	// Load row into shared memory
	int flatIndex = x + (width * y);
	sharedRow[x] = dev_framebuffer[flatIndex];

	__syncthreads();

	// Gaussian filter over row
	glm::vec3 result = sharedRow[x] * weight[0];
	for (int i = 1; i < 5; i++) {
		if (x + i < width) {
			result += sharedRow[x + i] * weight[i];
		}
		if (x - i >= 0) {
			result += sharedRow[x - i] * weight[i];
		}
	}
	
	dev_postprocess[flatIndex] = result;
}

__global__
void gaussianBlurHeight(int width, int height, glm::vec3 *dev_framebuffer, glm::vec3 *dev_postprocess) {
	extern __shared__ glm::vec3 sharedColumn[];
	int x = blockIdx.x;
	int y = threadIdx.x;

	float weight[5] = { 0.227027f, 0.1945946, 0.1216216, 0.054054, 0.016216 };

	if (x >= width || y >= height) {
		return;
	}

	// Load column into shared memory
	int flatIndex = (y * width) + x;
	sharedColumn[y] = dev_framebuffer[flatIndex];

	__syncthreads();

	// Gaussian filter over column
	glm::vec3 result = sharedColumn[y] * weight[0];
	for (int i = 1; i < 5; i++) {
		if (y + i < height) {
			result += sharedColumn[y + i] * weight[i];
		}
		if (y - i >= 0) {
			result += sharedColumn[y - i] * weight[i];
		}
	}

	dev_postprocess[flatIndex] = result;
}

/**
 * Perform rasterization.
 */
void rasterize(uchar4 *pbo, const glm::mat4 & MVP, const glm::mat4 & MV, const glm::mat3 MV_normal) {
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
		(height - 1) / blockSize2d.y + 1);

	// Execute your rasterization pipeline here
	// (See README for rasterization pipeline outline.)

	// Vertex Process & primitive assembly
	{
		curPrimitiveBeginId = 0;
		dim3 numThreadsPerBlock(128);

		auto it = mesh2PrimitivesMap.begin();
		auto itEnd = mesh2PrimitivesMap.end();

		for (; it != itEnd; ++it) {
			auto p = (it->second).begin();	// each primitive
			auto pEnd = (it->second).end();
			for (; p != pEnd; ++p) {
				dim3 numBlocksForVertices((p->numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
				dim3 numBlocksForIndices((p->numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);

				_vertexTransformAndAssembly << < numBlocksForVertices, numThreadsPerBlock >> >(p->numVertices, *p, MVP, MV, MV_normal, width, height);
				checkCUDAError("Vertex Processing");
				hipDeviceSynchronize();
				_primitiveAssembly << < numBlocksForIndices, numThreadsPerBlock >> >
					(p->numIndices, 
					curPrimitiveBeginId, 
					dev_primitives, 
					*p);
				checkCUDAError("Primitive Assembly");

				curPrimitiveBeginId += p->numPrimitives;
			}
		}

		checkCUDAError("Vertex Processing and Primitive Assembly");
	}
	
	hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));
	initDepth << <blockCount2d, blockSize2d >> >(width, height, dev_depth);
	
	// TODO: rasterize
	hipMemset(dev_mutex, 0, sizeof(int));
	dim3 numThreadsPerBlock(128);
	dim3 numBlocksPerPrimitive = (totalNumPrimitives + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x;
	_rasterize<<<numBlocksPerPrimitive, numThreadsPerBlock>>>(totalNumPrimitives, dev_primitives, dev_fragmentBuffer, dev_depth, dev_mutex, width, height);


    // Copy depthbuffer colors into framebuffer
	render << <blockCount2d, blockSize2d >> >(width, height, dev_fragmentBuffer, dev_framebuffer, dev_depth);
	checkCUDAError("fragment shader");

#ifdef GAUSSIAN_BLUR
	// Gaussian blur
	dim3 threadsPerBlock(width);
	dim3 blocksPerGrid(height);
	size_t sharedMemorySize(width * sizeof(glm::vec3));
	gaussianBlurWidth<<<blocksPerGrid, threadsPerBlock, sharedMemorySize>>>(width, height, dev_framebuffer, dev_postprocess);

	threadsPerBlock = height;
	blocksPerGrid = width;
	sharedMemorySize = height * sizeof(glm::vec3);
	gaussianBlurHeight<<<blocksPerGrid, threadsPerBlock, sharedMemorySize>>>(width, height, dev_framebuffer, dev_postprocess);

	glm::vec3 *temp = dev_postprocess;
	dev_postprocess = dev_framebuffer;
	dev_framebuffer = temp;
#endif

    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);
    checkCUDAError("copy render result to pbo");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {

    // deconstruct primitives attribute/indices device buffer

	auto it(mesh2PrimitivesMap.begin());
	auto itEnd(mesh2PrimitivesMap.end());
	for (; it != itEnd; ++it) {
		for (auto p = it->second.begin(); p != it->second.end(); ++p) {
			hipFree(p->dev_indices);
			hipFree(p->dev_position);
			hipFree(p->dev_normal);
			hipFree(p->dev_texcoord0);
			hipFree(p->dev_diffuseTex);

			hipFree(p->dev_verticesOut);

			
			//TODO: release other attributes and materials
		}
	}

	////////////

    hipFree(dev_primitives);
    dev_primitives = NULL;

	hipFree(dev_fragmentBuffer);
	dev_fragmentBuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

	hipFree(dev_postprocess);
	dev_postprocess = NULL;

	hipFree(dev_mutex);
	dev_mutex = NULL;

	hipFree(dev_depth);
	dev_depth = NULL;

    checkCUDAError("rasterize Free");
}
